#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>
#include <iostream>
#include <stdio.h>

#include "caffe/layers/knowledge_distillation_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {
	
template <typename Dtype>
__global__ void SoftmaxLossForwardGPU(const int nthreads,
          const Dtype* prob_data, const Dtype* soft_label, Dtype* loss_data, Dtype* counts) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    loss_data[index] = -soft_label[index] * (log(max(prob_data[index], Dtype(FLT_MIN)))-log(max(soft_label[index], Dtype(FLT_MIN))));
    counts[index] = 1;
  }
}

template <typename Dtype>
void KnowledgeDistillationLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // Both logits are divided by the temperature T.
  caffe_gpu_memcpy(bottom[0]->count() * sizeof(Dtype), bottom[0]->gpu_data(), s_logit_.mutable_gpu_data());
  caffe_gpu_scal(bottom[0]->count(), Dtype(1)/T, s_logit_.mutable_gpu_data());
  caffe_gpu_memcpy(bottom[1]->count() * sizeof(Dtype), bottom[1]->gpu_data(), t_logit_.mutable_gpu_data());
  caffe_gpu_scal(bottom[0]->count(), Dtype(1)/T, t_logit_.mutable_gpu_data());
  // The forward pass computes the softmax prob values.
  s_softmax_layer_->Forward(s_softmax_bottom_vec_, s_softmax_top_vec_);
  t_softmax_layer_->Forward(t_softmax_bottom_vec_, t_softmax_top_vec_);
  const Dtype* prob_data = s_prob_.gpu_data();
  const Dtype* soft_label = t_prob_.gpu_data();
  int dim = s_prob_.count() / outer_num_;
  int count = 0;
  Dtype loss = 0;
  int pos;
  // Compute KL divergence.
  if (bottom.size() == 2) { // label inputs or ignore_label are not given.
    count = outer_num_ * inner_num_;
	
	Dtype* loss_data = bottom[0]->mutable_gpu_diff();
	Dtype* counts = s_prob_.mutable_gpu_diff();
	const int nthreads = (outer_num_ - 1) * dim + (bottom[0]->shape(softmax_axis_) - 1) * inner_num_ + inner_num_;
	SoftmaxLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, prob_data, soft_label, loss_data, counts);
	caffe_gpu_asum(nthreads, loss_data, &loss);
  }

  top[0]->mutable_cpu_data()[0] = loss / get_normalizer(normalization_, count);
}

/*
template <typename Dtype>
__global__ void SoftmaxLossBackwardGPU(const int nthreads, const Dtype* top,
          const Dtype* label, Dtype* bottom_diff, Dtype* counts) {
  const int channels = dim / spatial_dim;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);

    if (has_ignore_label_ && label_value == ignore_label_) {
      for (int c = 0; c < channels; ++c) {
        bottom_diff[n * dim + c * spatial_dim + s] = 0;
      }
      counts[index] = 0;
    } else {
      bottom_diff[n * dim + label_value * spatial_dim + s] -= 1;
      counts[index] = 1;
    }
  }
}
*/

template <typename Dtype>
void KnowledgeDistillationLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* prob_data = s_prob_.gpu_data();
    caffe_gpu_memcpy(s_prob_.count() * sizeof(Dtype), prob_data, bottom_diff);
    const Dtype* soft_label = t_prob_.gpu_data();
    int dim = s_prob_.count() / outer_num_;
    int count = outer_num_ * inner_num_;
    // The gradients here are multiplied by T,
    // which is T^2 (as suggested in the paper) * 1/T (logits divided by T).
    caffe_gpu_axpby<Dtype>(outer_num_*dim, -T, soft_label, T, bottom_diff);

    // Scale gradient
    Dtype loss_weight = top[0]->cpu_diff()[0] /
                        get_normalizer(normalization_, count);
    caffe_gpu_scal(s_prob_.count(), loss_weight, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(KnowledgeDistillationLayer);

}  // namespace caffe
